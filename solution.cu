// do not modify this file
// call each kernel implemented in the kernel.cu
// generates timing info
// tests for functional verification

#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <wb.h>
#include "kernel.cu"
#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void histogram(unsigned int *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins, int kernel_version) {


 if (kernel_version == 0) {
  // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    histogram_global_kernel<<<gridDim, blockDim, num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  // Make sure bin values are not too large
  {
    dim3 blockDim(512);
    dim3 gridDim((num_bins + blockDim.x - 1) / blockDim.x);
    convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
 }
 else if (kernel_version==1) {
 // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    histogram_shared_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  // Make sure bin values are not too large
  {
    dim3 blockDim(512);
    dim3 gridDim((num_bins + blockDim.x - 1) / blockDim.x);
    convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
 }

else if (kernel_version==2) {
 // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    histogram_shared_accumulate_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  // Make sure bin values are not too large
  {
    dim3 blockDim(512);
    dim3 gridDim((num_bins + blockDim.x - 1) / blockDim.x);
    convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
 }


}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  int version; // kernel version global or shared 
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);
  
  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  CUDA_CHECK(hipMalloc((void **)&deviceInput,
                        inputLength * sizeof(unsigned int)));
  CUDA_CHECK(
      hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceInput, hostInput,
                        inputLength * sizeof(unsigned int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  // wbTime_start(Compute, "Performing CUDA computation");

  version = atoi(argv[5]); 
  hipEventRecord(astartEvent, 0);
  histogram(deviceInput, deviceBins, inputLength, NUM_BINS,version);
  // wbTime_stop(Compute, "Performing CUDA computation");

  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("\n");
  printf("Total compute time (ms) %f for version %d\n",aelapsedTime,version);
  printf("\n");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  // Verify correctness
  // -----------------------------------------------------
  printf ("running version %d\n", version);
  if (version == 0 )
     wbLog(TRACE, "Checking global memory only kernel");
  else if (version == 1) 
     wbLog(TRACE, "Launching shared memory kernel");
  else if (version == 2) 
     wbLog(TRACE, "Launching accumulator kernel");
  wbSolution(args, hostBins, NUM_BINS);

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  CUDA_CHECK(hipFree(deviceInput));
  CUDA_CHECK(hipFree(deviceBins));
  wbTime_stop(GPU, "Freeing GPU Memory");


  free(hostBins);
  free(hostInput);
  return 0;
}
